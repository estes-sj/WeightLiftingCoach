#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "detectNet.h"
#include "cudaUtility.h"
#include "cudaAlphaBlend.cuh"
#include <cstdlib>
#include <cmath>

#define MIN(a,b)	(a < b ? a : b)
#define MAX(a,b)	(a > b ? a : b)

template<typename T> inline __device__ __host__ T sqr(T x) 				    { return x*x; }

inline __device__ __host__ float dist2(float x0, float y0, float x1, float y1) { return sqr(x0-x1) + sqr(y0-y1); }
inline __device__ __host__ float dist(float x0, float y0, float x1, float y1)  { return sqrtf(dist2(x0,y0,x1,y1)); }



template<typename T>
__global__ void gpuDetectionOverlay( T* input, T* output, int width, int height, detectNet::Detection* detections, int numDetections, float4* colors ) 
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x >= width || y >= height )
		return;

	const int px_idx = y * width + x;
	T px = input[px_idx];
	
	const float fx = x;
	const float fy = y;
	
	for( int n=0; n < numDetections; n++ )
	{
		const detectNet::Detection det = detections[n];

		// check if this pixel is inside the bounding box
		if( fx >= det.Left && fx <= det.Right && fy >= det.Top && fy <= det.Bottom )
		{
			const float4 color = colors[det.ClassID];	

			const float alpha = color.w / 255.0f;
			const float ialph = 1.0f - alpha;

			px.x = alpha * color.x + ialph * px.x;
			px.y = alpha * color.y + ialph * px.y;
			px.z = alpha * color.z + ialph * px.z;
		}
	}
	
	output[px_idx] = px;	 
}

// Old Overlay
// template<typename T>
// __global__ void gpuDetectionOverlayBox( T* input, T* output, int imgWidth, int imgHeight, int x0, int y0, int boxWidth, int boxHeight, const float4 color ) 
// {
// 	const int box_x = blockIdx.x * blockDim.x + threadIdx.x;
// 	const int box_y = blockIdx.y * blockDim.y + threadIdx.y;

// 	if( box_x >= imgWidth || box_y >= imgHeight )
// 		return;

// 	const int x = box_x + x0;
// 	const int y = box_y + y0;

// 	if( x >= imgWidth || y >= imgHeight )
// 	return;

// 	T px = input[ y * imgWidth + x ];

// 	const float alpha = color.w / 255.0f;
// 	const float ialph = 1.0f - alpha;

// 	px.x = alpha * color.x + ialph * px.x;
// 	px.y = alpha * color.y + ialph * px.y;
// 	px.z = alpha * color.z + ialph * px.z;

// 	output[y * imgWidth + x] = px;
// }

// Line Distance Squard
inline __device__ float lineDistanceSquared(float x, float y, float x0, float y0, float x1, float y1)
{
	const float d = dist2(x0, y0, x1, y1);
	const float t = ( (x-x0) * (x1-x0) + (y-y0) * (y1-y0) ) / d;
	const float u = MAX( 0, MIN(1, t) );
	
	return dist2(x, y, x0 + u * (x1 - x0), y0 + u * (y1 - y0));
}

// New Overlay
template<typename T>
__global__ void gpuDrawLine( T* img, int imgWidth, int imgHeight, int offset_x, int offset_y, int x0, int y0, int x1, int y1, const float4 color, float line_width2 ) 
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x + offset_x;
	const int y = blockIdx.y * blockDim.y + threadIdx.x + offset_y;

	if( x >= imgWidth || y >= imgHeight )
		return;

	if( lineDistanceSquared(x, y, x0, y0, x1, y1) <= line_width2 )
	{
		const int idx = y * imgWidth + x;
		img[idx] = cudaAlphaBlend(img[idx], color );
	}

}

// // New cudaDrawLine 
// hipError_t cudaDrawLine( void* input, void* output, size_t width, size_t height, imageFormat format, int x1, int y1, int x2, int y2, const float4& color, float line_width )
// {
// 	if( !input || !output || width == 0 || height == 0 || line_width <= 0 )
// 		return hipErrorInvalidValue;
	
// 	// check for lines < 2 pixels in length
// 	if( dist(x1,y1,x2,y2) < 2.0 )
// 	{
// 		return hipSuccess;
// 	}
// 	// if the input and output images are different, copy the input to the output
// 	// this is because we only launch the kernel in the approximate area of the circle
// 	if( input != output )
// 		CUDA(hipMemcpy(output, input, imageFormatSize(format, width, height), hipMemcpyDeviceToDevice));
	

// 	// find a box around the line
// 	const int left = MIN(x1,x2) - line_width;
// 	const int right = MAX(x1,x2) + line_width;
// 	const int top = MIN(y1,y2) - line_width;
// 	const int bottom = MAX(y1,y2) + line_width;

// 	// launch kernel
// 	const dim3 blockDim(8, 8);
// 	const dim3 gridDim(iDivUp(right - left, blockDim.x), iDivUp(bottom - top, blockDim.y));

// 	#define LAUNCH_DRAW_LINE(type) \
// 		gpuDrawLine<type><<<gridDim, blockDim>>>((type*)output, width, height, left, top, x1, y1, x2, y2, color, line_width * line_width)
	
// 	if( format == IMAGE_RGB8 )
// 		LAUNCH_DRAW_LINE(uchar3);
// 	else if( format == IMAGE_RGBA8 )
// 		LAUNCH_DRAW_LINE(uchar4);
// 	else if( format == IMAGE_RGB32F )
// 		LAUNCH_DRAW_LINE(float3); 
// 	else if( format == IMAGE_RGBA32F )
// 		LAUNCH_DRAW_LINE(float4);
// 	else
// 	{
// 		return hipErrorInvalidValue;
// 	}
		
// 	return hipGetLastError();
// }

// Old Launch Overlay
// template<typename T>
// hipError_t launchDetectionOverlay( T* input, T* output, uint32_t width, uint32_t height, detectNet::Detection* detections, int numDetections, float4* colors )
// {
// 	if( !input || !output || width == 0 || height == 0 || !detections || numDetections == 0 || !colors )
// 		return hipErrorInvalidValue;
			
// 	// this assumes that the output already has the input image copied to it,
// 	// which if input != output, is done first by detectNet::Detect()
// 	for( int n=0; n < numDetections; n++ )
// 	{
// 		const int boxWidth = (int)detections[n].Width();
// 		const int boxHeight = (int)detections[n].Height();

// 		// launch kernel
// 		const dim3 blockDim(8, 8);
// 		const dim3 gridDim(iDivUp(boxWidth,blockDim.x), iDivUp(boxHeight,blockDim.y));

// 		gpuDetectionOverlayBox<T><<<gridDim, blockDim>>>(input, output, width, height, (int)detections[n].Left, (int)detections[n].Top, boxWidth, boxHeight, colors[detections[n].ClassID]); 
// 	}

// 	return hipGetLastError();
// }

// New Launch Detection
// template<typename T>
// hipError_t launchDetectionOverlay( T* input, T* output, uint32_t width, uint32_t height, int x0, int y0, int x1, int y1, const float4 color, const float line_width )
// {
// 	if( !input || !output || width == 0 || height == 0 )
// 		return hipErrorInvalidValue;

// 		// find a box around the line
// 		const int left = MIN(x0,x1) - line_width;
// 		const int right = MAX(x0,x1) + line_width;
// 		const int top = MIN(y0,y1) - line_width;
// 		const int bottom = MAX(y0,y1) + line_width;
	
// 		// launch kernel
// 		const dim3 blockDim(8, 8);
// 		const dim3 gridDim(iDivUp( right - left, blockDim.x ), iDivUp( bottom - top, blockDim.y ) );

// 		gpuDrawLine<T><<<gridDim, blockDim>>>(output, width, height, left, top, x0, y0, x1, y1, color, line_width * line_width);
// 		hipDeviceSynchronize();

// 	return hipGetLastError();
// }

// hipError_t cudaDetectionOverlay( void* input, void* output, uint32_t width, uint32_t height, imageFormat format, int x0, int y0, int x1, int y1, const float4& color, const float line_width )
// {
// 	if( format == IMAGE_RGB8 )
// 		return launchDetectionOverlay<uchar3>((uchar3*)input, (uchar3*)output, width, height, x0, y0, x1, y1, color, line_width); 
// 	else if( format == IMAGE_RGBA8 )
// 		return launchDetectionOverlay<uchar4>((uchar4*)input, (uchar4*)output, width, height, x0, y0, x1, y1, color, line_width);  
// 	else if( format == IMAGE_RGB32F )
// 		return launchDetectionOverlay<float3>((float3*)input, (float3*)output, width, height, x0, y0, x1, y1, color, line_width);  
// 	else if( format == IMAGE_RGBA32F )
// 		return launchDetectionOverlay<float4>((float4*)input, (float4*)output, width, height, x0, y0, x1, y1, color, line_width); 
// 	else
// 		return hipErrorInvalidValue;
// }

